#include "hip/hip_runtime.h"
#include "./spada_lib.h"
#define WINDOW_HEIGHT 4
#define WINDOW_WIDTH 8


#define DELLEXPORT extern "C" __declspec(dllexport)
using namespace std;


__global__  void spgemmByAdaptiveWindow_4_8(const CSRMatrix* A, const CSRMatrix* B, CSRMatrix* C)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	if (row < A->rows)
	{
		for(int i = 0; i < ceil((float)A->rows / WINDOW_HEIGHT); i++)
		{
			 if (row + i * WINDOW_HEIGHT >= A->rows) 
			 {
			 	break;
			 }
			__shared__ int indexHolder[WINDOW_HEIGHT];
			for(int j = 0; j < WINDOW_HEIGHT; j++)
			{
				indexHolder[j] = 0;
			}

			for(int j = 0; j < ceil((float)A->cols / WINDOW_WIDTH); j++)
			{
				int index = A->rowPtr[row + i * WINDOW_HEIGHT] + col + j * WINDOW_WIDTH;

				if (index >= A->rowPtr[row + i * WINDOW_HEIGHT + 1]) 
				{
					break;
				}
		
				int idxA = A->colInd[index];
				 if (idxA >= A->cols) 
				 {
				 	break;
				 }
				for(int k = B->rowPtr[idxA]; k < B->rowPtr[idxA+1]; k++)
				{
					int idxC = C->rowPtr[row + i * WINDOW_HEIGHT] + indexHolder[row];
					indexHolder[row]++;
					 if(idxC >= C->nnz)
					 {
					 	break;
					 }

					atomicAdd(&C->values[idxC], A->values[index] * B->values[k]);
					C->colInd[idxC] = B->colInd[k];
				}


			}
		}
	}
}

DELLEXPORT float spada_4_8(const bool isOne, const char* fileName, COOMatrix** h_coo_a, COOMatrix** h_coo_b, CSRMatrix** h_csr_c)
{

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	try {
		readFile(fileName, *h_coo_a, isOne);
	}
	catch (exception& e) {
		cout << "An error occurred: " << e.what() << endl;
	}


	CSRMatrix csrA, csrB, csrC;

	transpose(**h_coo_a, *h_coo_b);
	convertCOOToCSR(*h_coo_a, &csrA);
	convertCOOToCSR(*h_coo_b, &csrB);

	CSRMatrix* d_A, *d_B, *d_C;

	hipMalloc((void**)&d_A, sizeof(CSRMatrix));
	hipMalloc((void**)&d_B, sizeof(CSRMatrix));
	hipMalloc((void**)&d_C, sizeof(CSRMatrix));

	int* d_A_rowPtr, *d_A_colInd, *d_B_rowPtr, *d_B_colInd, *d_C_rowPtr, *d_C_colInd;
	float* d_A_values, *d_B_values, *d_C_values;

	hipMalloc((void**)&d_A_rowPtr, (csrA.rows + 1) * sizeof(int));
	hipMalloc((void**)&d_A_colInd, csrA.nnz * sizeof(int));
	hipMalloc((void**)&d_A_values, csrA.nnz * sizeof(float));
	hipMemcpy(d_A_rowPtr, csrA.rowPtr, (csrA.rows + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_A_colInd, csrA.colInd, csrA.nnz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_A_values, csrA.values, csrA.nnz * sizeof(float), hipMemcpyHostToDevice);

	CSRMatrix* d_A_struct = new CSRMatrix;
	d_A_struct->rowPtr = d_A_rowPtr;
	d_A_struct->colInd = d_A_colInd;
	d_A_struct->values = d_A_values;
	d_A_struct->rows = csrA.rows;
	d_A_struct->cols = csrA.cols;
	d_A_struct->nnz = csrA.nnz;
	d_A_struct->nnzRow = csrA.nnzRow;

	hipMemcpy(d_A, d_A_struct, sizeof(CSRMatrix), hipMemcpyHostToDevice);

	hipMalloc((void**)&d_B_rowPtr, (csrB.rows + 1) * sizeof(int));
	hipMalloc((void**)&d_B_colInd, csrB.nnz * sizeof(int));
	hipMalloc((void**)&d_B_values, csrB.nnz * sizeof(float));
	hipMemcpy(d_B_rowPtr, csrB.rowPtr, (csrB.rows + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_B_colInd, csrB.colInd, csrB.nnz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_B_values, csrB.values, csrB.nnz * sizeof(float), hipMemcpyHostToDevice);
	
	CSRMatrix* d_B_struct = new CSRMatrix;
	d_B_struct->rowPtr = d_B_rowPtr;
	d_B_struct->colInd = d_B_colInd;
	d_B_struct->values = d_B_values;
	d_B_struct->rows = csrB.rows;
	d_B_struct->cols = csrB.cols;
	d_B_struct->nnz = csrB.nnz;
	d_B_struct->nnzRow = csrB.nnzRow;
	
	hipMemcpy(d_B, d_B_struct, sizeof(CSRMatrix), hipMemcpyHostToDevice);


	csrC.rows = csrA.rows;
	csrC.cols = csrB.cols;
	csrC.rowPtr = new int[csrC.rows + 1];
	csrC.nnz = 0;
	csrC.nnzRow = 0;

	int* d_rowPtrC, *d_nnzC;
	hipMalloc((void**)&d_rowPtrC, (csrC.rows + 1) * sizeof(int));
	hipMalloc((void**)&d_nnzC, sizeof(int));
	hipMemset(d_nnzC, 0, sizeof(int));
	
	hipMemset(d_rowPtrC, 0, (csrC.rows + 1) * sizeof(int));
	
	
	
	int THREADS = 1024;
	int BLOCKS = ceil((float)csrC.rows / THREADS);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	preciseSizePredictionKernel<<<BLOCKS, THREADS>>>(d_A, d_B, d_rowPtrC, d_nnzC);

	hipDeviceSynchronize();

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << "Time for precise size prediction: " << milliseconds << "ms" << endl;
	

	hipMemcpy(csrC.rowPtr, d_rowPtrC, (csrC.rows + 1) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&csrC.nnz, d_nnzC, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_rowPtrC);
	hipFree(d_nnzC);

	for(int i = 1; i < csrC.rows + 1; i++)
	{
		csrC.rowPtr[i] = csrC.rowPtr[i] + csrC.rowPtr[i-1];
	}	

	csrC.colInd = new int[csrC.nnz];
	csrC.values = new float[csrC.nnz];

	for(int i = 0; i < csrC.nnz; i++)
	{
		csrC.values[i] = 0;
		csrC.colInd[i] = -1;
	}


	hipMalloc((void**)&d_C_rowPtr, (csrC.rows + 1) * sizeof(int));
	hipMalloc((void**)&d_C_colInd, csrC.nnz * sizeof(int));
	hipMalloc((void**)&d_C_values, csrC.nnz * sizeof(float));
	hipMemcpy(d_C_rowPtr, csrC.rowPtr, (csrC.rows + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_C_colInd, csrC.colInd, csrC.nnz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_C_values, csrC.values, csrC.nnz * sizeof(float), hipMemcpyHostToDevice);

	CSRMatrix* d_C_struct = new CSRMatrix;
	d_C_struct->rowPtr = d_C_rowPtr;
	d_C_struct->colInd = d_C_colInd;
	d_C_struct->values = d_C_values;
	d_C_struct->rows = csrC.rows;
	d_C_struct->cols = csrC.cols;
	d_C_struct->nnz = csrC.nnz;
	d_C_struct->nnzRow = csrC.nnzRow;

	hipMemcpy(d_C, d_C_struct, sizeof(CSRMatrix), hipMemcpyHostToDevice);
	

	hipEventCreate(&start);
	hipEventCreate(&stop);


	dim3 dimGrid(1, 1);
	dim3 dimBlock(WINDOW_HEIGHT, WINDOW_WIDTH);
	hipEventRecord(start);
	spgemmByAdaptiveWindow_4_8<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	
	hipEventRecord(stop);

	hipEventSynchronize(stop);

	float kernelTime = 0;
	hipEventElapsedTime(&kernelTime, start, stop);
	cout << "Time for kernel execution: " << kernelTime << "ms" << endl;


	hipEventRecord(start);
	shrinkResultMatrix<<<BLOCKS, THREADS>>>(d_C);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << "Time for shrinking result matrix: " << milliseconds << "ms" << endl;

	int* d_result_rowPtr, *d_result_colInd;
	float* d_result_values;
	

	int resultNnz;
	hipMemcpy(&resultNnz, &d_C->nnz, sizeof(int), hipMemcpyDeviceToHost);
	int* temp_row;
	temp_row = new int[csrC.rows + 1];


	hipMalloc((void**)&d_result_rowPtr, (csrC.rows + 1) * sizeof(int));
	hipMalloc((void**)&d_result_colInd, resultNnz * sizeof(int));
	hipMalloc((void**)&d_result_values, resultNnz * sizeof(float));
	hipMemset(d_result_rowPtr, 0, (csrC.rows + 1) * sizeof(int));

	hipEventRecord(start);
	rowPtrRecalculation<<<BLOCKS, THREADS>>>(d_C, d_result_rowPtr);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << "Time for recalculating rowPtr: " << milliseconds << "ms" << endl;


	hipMemcpy(temp_row, d_result_rowPtr, (csrC.rows + 1) * sizeof(int), hipMemcpyDeviceToHost);
	for(int i = 1; i < csrC.rows + 1; i++)
	{
		temp_row[i] = temp_row[i] + temp_row[i-1];
	}
	hipMemcpy(d_result_rowPtr, temp_row, (csrC.rows + 1) * sizeof(int), hipMemcpyHostToDevice);

	hipEventRecord(start);
	createResultMatrix<<<BLOCKS, THREADS>>>(d_C, d_result_rowPtr, d_result_colInd, d_result_values);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << "Time for creating result matrix: " << milliseconds << "ms" << endl;


	hipMemcpy(csrC.rowPtr, d_result_rowPtr, (csrC.rows + 1) * sizeof(int), hipMemcpyDeviceToHost);

	delete[] csrC.colInd;
	delete[] csrC.values;
	csrC.colInd = new int[resultNnz];
	csrC.values = new float[resultNnz];

	hipMemcpy(csrC.colInd, d_result_colInd, resultNnz * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(csrC.values, d_result_values, resultNnz * sizeof(float), hipMemcpyDeviceToHost);
	csrC.nnz = resultNnz;
	
	copyCSRMatrix(csrC, *h_csr_c);

	cout << endl;

	hipFree(d_A_rowPtr);
	hipFree(d_A_colInd);
	hipFree(d_A_values);
	hipFree(d_B_rowPtr);
	hipFree(d_B_colInd);
	hipFree(d_B_values);
	hipFree(d_C_rowPtr);
	hipFree(d_C_colInd);
	hipFree(d_C_values);
	hipFree(d_result_rowPtr);
	hipFree(d_result_colInd);
	hipFree(d_result_values);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	delete[] csrA.rowPtr;
	delete[] csrA.colInd;
	delete[] csrA.values;
	delete[] csrB.rowPtr;
	delete[] csrB.colInd;
	delete[] csrB.values;
	delete[] csrC.rowPtr;
	delete[] csrC.colInd;
	delete[] csrC.values;
	delete[] temp_row;

	delete d_A_struct;
	delete d_B_struct;
	delete d_C_struct;

	return kernelTime;
}
